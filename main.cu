#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <unistd.h>
#include <sys/time.h>
#include <map>
#include <functional>
#include "seq.hpp"
#include "cudaThreads.cuh"
#include "cudaBlocks.cuh"
#include "cudaThreadsShared.cuh"

std::map<uint8_t, std::string> VersionsMap =
    {
        {0, "SEQ"},
        {1, "CUDA_THREADS"},
        {2, "CUDA_BLOCKS"},
        {3, "CUDA_THREADS_SHARED"},
        {4, "CUDA_THREADS_GEN"},
        {5, "CUDA_BLOCKS_GEN"},
        {6, "CUDA_THREADS_SHARED_GEN"},
        {7, "CUDA_BLOCKS_GEN_GRAPH"},
        {8, "CUDA_THREADS_GEN_GRAPH"},
        {9, "CUDA_THREADS_SHARED_GEN_GRAPH"},
        {10, "CUDA_BLOCKS_GEN_STREAMS"},
        {11, "CUDA_BLOCKS_GEN_GRAPH_STREAMS"},
        {12, "ALL_GEN"},
        {13, "ALL_GEN_GRAPH"}};

int main(int argc, char **argv)
{
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // Needed to use hipStreamSynchronize instead of hipDeviceSynchronize which is slower
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, device);
    std::cout << "Cooperative launch support: " << supportsCoopLaunch << std::endl; // 1 if supported, 0 otherwise (if 0, use atomic counter instead of cooperative launch)

    printf("Max grid size of dimesion x: %d bytes\n", prop.maxGridSize[0]); // change macro of MAX_BLOCKS if necessary
    printf("Max threads per block: %d bytes\n", prop.maxThreadsPerBlock); // change macro of MAX_THREADS_PER_BLOCK if necessary
    printf("Max shared memory per block: %ld bytes\n", prop.sharedMemPerBlock); // change macro of MAX_SHARED_MEMORY if necessary

    uint8_t version;
    size_t n;
    uint32_t k, blocks, threadsPerBlock;
    switch (argc)
    {
    case 1:
        std::cout << "Usage: " << argv[0] << " <version> <n> <k> <number of blocks> <number of threads per block>" << std::endl;
        return 0;
    case 2:
        version = atoi(argv[1]);
        std::cout << "You need to specify the array dimension" << std::endl;
        return 0;
    case 3:
        version = atoi(argv[1]);
        n = atoi(argv[2]);
        if (version > 1)
        {
            std::cout << "You need to specify the number of blocks for this version" << std::endl;
            return 0;
        }
        std::cout << "Num of iterations not specified. Setting k = 1" << std::endl;
        k = 1;
        break;
    case 4:
        version = atoi(argv[1]);
        if (version > 1)
        {
            std::cout << "You need to specify the number of blocks for this version" << std::endl;
            return 0;
        }
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        break;
    case 5:
        version = atoi(argv[1]);
        if (version == 3)
        {
            std::cout << "You need to specify the number of threads for this version" << std::endl;
            return 0;
        }
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        blocks = atoi(argv[4]);
        break;
    case 6:
        version = atoi(argv[1]);
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        blocks = atoi(argv[4]);
        threadsPerBlock = atoi(argv[5]);
        if (version > 2 && threadsPerBlock > MAX_THREADS_PER_BLOCK)
        {
            std::cout << "Max number of threads per block is " << MAX_THREADS_PER_BLOCK << std::endl;
            return 0;
        }
        break;
    }

    srand(time(NULL));
    std::vector<uint8_t> in(n * n);
    std::vector<uint8_t> out(n * n);
    for (size_t i = 0; i < in.size(); i++)
    {
        in[i] = rand() % 2;
        // if (i != 0 && i % ((size_t)sqrt(in.size())) == 0)
        //     std::cout << std::endl;
        // std::cout << unsigned(in[i]) << " ";
    }
    // std::cout << std::endl;

    std::vector<uint8_t> in_copy(n * n);
    in_copy = in;
    std::vector<uint8_t> outSeq(n * n);

    std::map<uint8_t, std::function<void()>> run = {
        {0, [&]()
         { isingSeq(out, in, k); }},
        {1, [&]()
         { isingCuda(out, in, k); }},
        {2, [&]()
         { isingCuda(out, in, k, blocks); }},
        {3, [&]()
         { isingCuda(out, in, k, blocks, threadsPerBlock); }},
        {4, [&]()
         { isingCudaGen(out, in, k); }},
        {5, [&]()
         { isingCudaGen(out, in, k, blocks); }},
        {6, [&]()
         { isingCudaGen(out, in, k, blocks, threadsPerBlock); }},
        {7, [&]()
         { isingCudaGenGraph(out, in, k, blocks); }},
        {8, [&]()
         { isingCudaGenGraph(out, in, k); }},
        {9, [&]()
         { isingCudaGenGraph(out, in, k, blocks, threadsPerBlock); }},
        {10, [&]()
         { isingCudaGenStreams(out, in, k, blocks); }},
        {11, [&]()
         { isingCudaGenGraphStreams(out, in, k, blocks); }}};

    struct timeval start, end;

    std::cout << "Running version " << VersionsMap[version] << std::endl;

    if (version < 12)
    {
        // run the version specified
        gettimeofday(&start, NULL);
        run[version]();
        gettimeofday(&end, NULL);

        if (VersionsMap[version] == "SEQ")
        {
            std::cout << "Time: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
            return 0;
        }

        hipDeviceSynchronize();

        in = in_copy; // reset input

        isingSeq(outSeq, in, k);
        // std::cout << "out:" << std::endl;
        // for (size_t i = 0; i < out.size(); i++)
        // {
        //     if (i != 0 && i % ((size_t)sqrt(out.size())) == 0)
        //         std::cout << std::endl;
        //     std::cout << unsigned(out[i]) << " ";
        // }
        // std::cout << std::endl;
        std::cout << "Seq and Cuda are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;

        return 0;
    }

    // run all versions

    isingSeq(outSeq, in, k);

    hipDeviceSynchronize();
    in = in_copy;

    if (VersionsMap[version] == "ALL_GEN")
    {
        gettimeofday(&start, NULL);
        isingCudaGen(out, in, k, blocks);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda blocks gen are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time blocks gen: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        gettimeofday(&start, NULL);
        isingCudaGenStreams(out, in, k, blocks);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda blocks gen streams are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time blocks gen streams: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        gettimeofday(&start, NULL);
        isingCudaGen(out, in, k);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda threads gen are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time threads gen: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        gettimeofday(&start, NULL);
        isingCudaGen(out, in, k, blocks, threadsPerBlock);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda threads shared gen are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time threads shared gen: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
    }
    else
    {
        gettimeofday(&start, NULL);
        isingCudaGenGraph(out, in, k, blocks);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda blocks gen graph are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time blocks gen graph: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        gettimeofday(&start, NULL);
        isingCudaGenGraphStreams(out, in, k, blocks);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda blocks gen graph streams are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time blocks gen graph streams: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        gettimeofday(&start, NULL);
        isingCudaGenGraph(out, in, k);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda threads gen graph are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time threads gen graph: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        gettimeofday(&start, NULL);
        isingCudaGenGraph(out, in, k, blocks, threadsPerBlock);
        gettimeofday(&end, NULL);
        std::cout << "Seq and Cuda threads shared gen graph are equal: " << (out == outSeq) << std::endl;
        std::cout << "Time threads shared gen graph: " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " us" << std::endl;
        hipDeviceSynchronize();
    }

    return 0;
}