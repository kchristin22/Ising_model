#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <unistd.h>
#include <sys/time.h>
#include <map>
#include <functional>
#include "seq.hpp"
#include "cudaThreads.cuh"
#include "cudaBlocks.cuh"
#include "cudaThreadsShared.cuh"

#define ANKERL_NANOBENCH_IMPLEMENT
#include "nanobench.h"

std::map<uint8_t, std::string> VersionsMap =
    {
        {0, "SEQ"},
        {1, "CUDA_THREADS"},
        {2, "CUDA_BLOCKS"},
        {3, "CUDA_THREADS_SHARED"},
        {4, "CUDA_THREADS_GEN"},
        {5, "CUDA_BLOCKS_GEN"},
        {6, "CUDA_THREADS_SHARED_GEN"},
        {7, "CUDA_BLOCKS_GEN_GRAPH"},
        {8, "CUDA_THREADS_GEN_GRAPH"},
        {9, "CUDA_THREADS_SHARED_GEN_GRAPH"},
        {10, "CUDA_BLOCKS_GEN_STREAMS"},
        {11, "CUDA_BLOCKS_GEN_GRAPH_STREAMS"},
        {12, "ALL_GEN"},
        {13, "ALL_GEN_GRAPH"}};

int main(int argc, char **argv)
{
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // Needed to use hipStreamSynchronize instead of hipDeviceSynchronize which is slower
    std::cout << "hipDeviceScheduleBlockingSync flag: " << prop.kernelExecTimeoutEnabled << std::endl;

    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, device);
    std::cout << "Cooperative launch support: " << supportsCoopLaunch << std::endl;

    printf("Max grid size of dimesion x: %d bytes\n", prop.maxGridSize[0]); // change macro of MAX_BLOCKS if necessary

    uint8_t version;
    size_t n;
    uint32_t k, blocks, threadsPerBlock;
    switch (argc)
    {
    case 1:
        std::cout << "Usage: " << argv[0] << " <version> <n> <k> <number of blocks> <number of threads per block>" << std::endl;
        return 0;
    case 2:
        version = atoi(argv[1]);
        std::cout << "You need to specify the array dimension" << std::endl;
        return 0;
    case 3:
        version = atoi(argv[1]);
        n = atoi(argv[2]);
        if (version > 1)
        {
            std::cout << "You need to specify the number of blocks for this version" << std::endl;
            return 0;
        }
        std::cout << "Num of iterations not specified. Setting k = 1" << std::endl;
        k = 1;
        break;
    case 4:
        version = atoi(argv[1]);
        if (version > 1)
        {
            std::cout << "You need to specify the number of blocks for this version" << std::endl;
            return 0;
        }
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        break;
    case 5:
        version = atoi(argv[1]);
        if (version == 3)
        {
            std::cout << "You need to specify the number of threads for this version" << std::endl;
            return 0;
        }
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        blocks = atoi(argv[4]);
        break;
    case 6:
        version = atoi(argv[1]);
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        blocks = atoi(argv[4]);
        threadsPerBlock = atoi(argv[5]);
        if (version > 2 && threadsPerBlock > MAX_THREADS_PER_BLOCK)
        {
            std::cout << "Max number of threads per block is " << MAX_THREADS_PER_BLOCK << std::endl;
            return 0;
        }
        break;
    }

    srand(time(NULL));
    std::vector<uint8_t> in(n * n);
    std::vector<uint8_t> out(n * n);
    for (size_t i = 0; i < in.size(); i++)
    {
        in[i] = rand() % 2;
        // if (i != 0 && i % ((size_t)sqrt(in.size())) == 0)
        //     std::cout << std::endl;
        // std::cout << unsigned(in[i]) << " ";
    }
    // std::cout << std::endl;

    std::vector<uint8_t> in_copy(n * n);
    in_copy = in;
    std::vector<uint8_t> outSeq(n * n);

    std::map<uint8_t, std::function<void()>> run = {
        {0, [&]()
         { isingSeq(out, in, k); }},
        {1, [&]()
         { isingCuda(out, in, k); }},
        {2, [&]()
         { isingCuda(out, in, k, blocks); }},
        {3, [&]()
         { isingCuda(out, in, k, blocks, threadsPerBlock); }},
        {4, [&]()
         { isingCudaGen(out, in, k); }},
        {5, [&]()
         { isingCudaGen(out, in, k, blocks); }},
        {6, [&]()
         { isingCudaGen(out, in, k, blocks, threadsPerBlock); }},
        {7, [&]()
         { isingCudaGenGraph(out, in, k, blocks); }},
        {8, [&]()
         { isingCudaGenGraph(out, in, k); }},
        {9, [&]()
         { isingCudaGenGraph(out, in, k, blocks, threadsPerBlock); }},
        {10, [&]()
         { isingCudaGenStreams(out, in, k, blocks); }},
        {11, [&]()
         { isingCudaGenGraphStreams(out, in, k, blocks); }}};

    std::cout << "Running version " << VersionsMap[version] << std::endl;

    if (version < 12)
    {
        char *filename = new char[17];
        sprintf(filename, "version_%d.json", version);
        std::fstream file(filename, std::ios::out);

        // run and benchmark the version specified
        ankerl::nanobench::Bench()
            .minEpochIterations(100)
            .epochs(5)
            .run(filename, [&]
                 { run[version](); })
            .render(ankerl::nanobench::templates::pyperf(), file);

        if (VersionsMap[version] == "SEQ")
            return 0;

        hipDeviceSynchronize();

        in = in_copy; // reset input

        isingSeq(outSeq, in, k);
        // std::cout << "out:" << std::endl;
        // for (size_t i = 0; i < out.size(); i++)
        // {
        //     if (i != 0 && i % ((size_t)sqrt(out.size())) == 0)
        //         std::cout << std::endl;
        //     std::cout << unsigned(out[i]) << " ";
        // }
        // std::cout << std::endl;
        std::cout << "Seq and Cuda are equal: " << (out == outSeq) << std::endl;

        return 0;
    }

    // run all versions

    isingSeq(outSeq, in, k);

    hipDeviceSynchronize();
    in = in_copy;

    if (VersionsMap[version] == "ALL_GEN")
    {
        std::fstream blocksFile("blocks_gen", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(10)
            .epochs(5)
            .run("blocks_gen", [&]
                 { isingCudaGen(out, in, k, blocks); })
            .render(ankerl::nanobench::templates::pyperf(), blocksFile);

        std::cout << "Seq and Cuda blocks gen are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        std::fstream blocksStreamsFile("blocks_gen_streams", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(10)
            .epochs(5)
            .run("blocks_gen_streams", [&]
                 { isingCudaGenStreams(out, in, k, blocks); })
            .render(ankerl::nanobench::templates::pyperf(), blocksStreamsFile);

        std::cout << "Seq and Cuda blocks gen streams are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        std::fstream threadsFile("threads_gen", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(100)
            .epochs(5)
            .run("threads_gen", [&]
                 { isingCudaGen(out, in, k); })
            .render(ankerl::nanobench::templates::pyperf(), threadsFile);

        std::cout << "Seq and Cuda threads gen are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        std::fstream threadsSharedFile("threads_shared_gen", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(100)
            .epochs(5)
            .run("threads_shared_gen", [&]
                 { isingCudaGen(out, in, k, blocks, threadsPerBlock); })
            .render(ankerl::nanobench::templates::pyperf(), threadsSharedFile);

        std::cout << "Seq and Cuda threads shared gen are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
    }
    else
    {
        std::fstream blocksFile("blocks_gen_graph", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(10)
            .epochs(5)
            .run("blocks_gen_graph", [&]
                 { isingCudaGenGraph(out, in, k, blocks); })
            .render(ankerl::nanobench::templates::pyperf(), blocksFile);

        std::cout << "Seq and Cuda blocks gen graph are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        std::fstream blocksStreamsFile("blocks_gen_graph_streams", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(10)
            .epochs(5)
            .run("blocks_gen_graph_streams", [&]
                 { isingCudaGenGraphStreams(out, in, k, blocks); })
            .render(ankerl::nanobench::templates::pyperf(), blocksStreamsFile);

        std::cout << "Seq and Cuda blocks gen graph streams are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        std::fstream threadsFile("threads_gen_graph", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(100)
            .epochs(5)
            .run("threads_gen_graph", [&]
                 { isingCudaGenGraph(out, in, k); })
            .render(ankerl::nanobench::templates::pyperf(), threadsFile);

        std::cout << "Seq and Cuda threads gen graph are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
        in = in_copy;

        std::fstream threadsSharedFile("threads_shared_gen_graph", std::ios::out);

        ankerl::nanobench::Bench()
            .minEpochIterations(100)
            .epochs(5)
            .run("threads_shared_gen_graph", [&]
                 { isingCudaGenGraph(out, in, k, blocks, threadsPerBlock); })
            .render(ankerl::nanobench::templates::pyperf(), threadsSharedFile);

        std::cout << "Seq and Cuda threads shared gen graph are equal: " << (out == outSeq) << std::endl;
        hipDeviceSynchronize();
    }

    return 0;
}