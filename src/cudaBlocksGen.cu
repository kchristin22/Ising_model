#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "cudaBlocks.cuh"

__device__ inline size_t upOffset(size_t n, size_t i)
{
    return (i - n + n * n) % (n * n);
}

__device__ inline size_t downOffset(size_t n, size_t i)
{
    return (i + n) % (n * n);
}

__device__ inline size_t leftOffset(size_t n, size_t i)
{
    return i / n * n + (i - 1 + n) % n;
}

__device__ inline size_t rightOffset(size_t n, size_t i)
{
    return i / n * n + (i + 1) % n;
}

__device__ inline size_t centerOffset(size_t n, size_t i)
{
    return i;
}

__device__ funcP upP = upOffset;
__device__ funcP downP = downOffset;
__device__ funcP leftP = leftOffset;
__device__ funcP rightP = rightOffset;
__device__ funcP centerP = centerOffset;

// auxiliary function for atomicAdd on uint8_t
__device__ static inline char atomicAdd(char *address, char val)
{
    // offset, in bytes, of the char* address within the 32-bit address of the space that overlaps it
    size_t long_address_modulo = (size_t)address & 3;
    // the 32-bit address that overlaps the same memory
    auto *base_address = (unsigned int *)((char *)address - long_address_modulo);
    // A 0x3210 selector in __byte_perm will simply select all four bytes in the first argument in the same order.
    // The "4" signifies the position where the first byte of the second argument will end up in the output.
    unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
    // for selecting bytes within a 32-bit chunk that correspond to the char* address (relative to base_address)
    unsigned int selector = selectors[long_address_modulo];
    unsigned int long_old, long_assumed, long_val, replacement;

    long_old = *base_address;

    do
    {
        long_assumed = long_old;
        // replace bits in long_old that pertain to the char address with those from val
        long_val = __byte_perm(long_old, 0, long_address_modulo) + val;
        replacement = __byte_perm(long_old, long_val, selector);
        long_old = atomicCAS(base_address, long_assumed, replacement);
    } while (long_old != long_assumed);
    return __byte_perm(long_old, 0, long_address_modulo);
}

__global__ void addValue(uint8_t *out, const uint8_t *in, const size_t n, const uint32_t blockChunk)
{
    size_t n2 = n * n;
    size_t start = blockIdx.x * blockChunk;
    size_t end = start + blockChunk;
    if (end > n2)
        end = n2;

    for (size_t i = start; i < end; i++)
    {
        size_t up = (i - n + n2) % n2;
        size_t down = (i + n) % n2;
        size_t row = i / n;
        size_t left = row * n + (i - 1 + n) % n;
        size_t right = row * n + (i + 1) % n;
        out[i] = (in[i] + in[up] + in[down] + in[left] + in[right]) > 2;
    }
}

__global__ void addValueStreams(uint8_t *out, const uint8_t *in, const size_t n, const uint32_t blockChunk, const funcP calcOffset)
{
    size_t n2 = n * n;
    size_t start = blockIdx.x * blockChunk;
    size_t end = start + blockChunk;
    if (end > n2)
        end = n2;

    for (size_t i = start; i < end; i++)
    {
        if (in[i] == 0)
            continue;                        // skip if the value is 0 (no need to add it to the neighbors)
        size_t offset = (*calcOffset)(n, i); // alocate memory for calcOffset
        atomicAdd((char *)&out[offset], 1);
        __threadfence();
    }
}

__global__ void assignClearValue(uint8_t *out, uint8_t *in, const size_t n, const uint32_t blockChunk)
{
    size_t n2 = n * n;
    size_t start = blockIdx.x * blockChunk;
    size_t end = start + blockChunk;
    if (end > n2)
        end = n2;

    for (size_t i = start; i < end; i++)
        in[i] = out[i]; // swap the pointers and assign the final value of this iteration
}

__global__ void assignClearValueStreams(uint8_t *out, uint8_t *in, const size_t n, const uint32_t blockChunk)
{
    size_t n2 = n * n;
    size_t start = blockIdx.x * blockChunk;
    size_t end = start + blockChunk;
    if (end > n2)
        end = n2;

    for (size_t i = start; i < end; i++)
    {
        in[i] = out[i] > 2; // swap the pointers and assign the final value of this iteration
        out[i] = 0;         // clear the output for the next iteration
    }
}

void isingCudaGen(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMallocAsync((void **)&d_in, n2 * sizeof(uint8_t), 0); // not async as we also need to copy the input
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMallocAsync((void **)&d_out, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the input from CPU to the device
    error = hipMemcpyAsync(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk;
    if (blocks > n2)
    {
        std::cout << "No need for that many blocks. Using " << n2 << " blocks" << std::endl;
        blocks = n2;
        blockChunk = 1;
    }
    else
    {
        blockChunk = n2 / blocks;                         // number of elements each block will process
        blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                          // processed per block will be as expected
    }

    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks. Using " << MAX_BLOCKS << " blocks" << std::endl;
        blocks = MAX_BLOCKS;
        blockChunk = (uint32_t)ceil((double)n2 / blocks);
    }

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);

    for (size_t iter = 0; iter < k; iter++)
    {
        // Launch the kernel
        addValue<<<blocks, 1>>>(d_out, d_in, n, blockChunk);
        error = hipGetLastError(); // Since no error was returned from all the previous cuda calls,
                                    // the last error must be from the kernel launches

        if (error != hipSuccess)
        {
            fprintf(stderr, "Add Kernel launch failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
        }

        assignClearValue<<<blocks, 1>>>(d_out, d_in, n, blockChunk); // no sync needed because default stream is synchronous to the others
        error = hipGetLastError();                                  // Since no error was returned from all the previous cuda calls,
                                                                     // the last error must be from the kernel launches
        if (error != hipSuccess)
        {
            fprintf(stderr, "Assign Kernel launch failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
        }
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipStreamSynchronize(0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the output back to the host
    error = hipMemcpyAsync(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // output is in d_in
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFreeAsync(d_in, 0);
    hipFreeAsync(d_out, 0);
}

void isingCudaGenGraph(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMallocAsync((void **)&d_in, n2 * sizeof(uint8_t), 0); // not async as we also need to copy the input
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMallocAsync((void **)&d_out, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the input from CPU to the device
    error = hipMemcpyAsync(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk;
    if (blocks > n2)
    {
        std::cout << "No need for that many blocks. Using " << n2 << " blocks" << std::endl;
        blocks = n2;
        blockChunk = 1;
    }
    else
    {
        blockChunk = n2 / blocks;                         // number of elements each block will process
        blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                          // processed per block will be as expected
    }

    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks. Using " << MAX_BLOCKS << " blocks" << std::endl;
        blocks = MAX_BLOCKS;
        blockChunk = (uint32_t)ceil((double)n2 / blocks);
    }

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);

    hipStream_t stream; // stream capture is only supported on non-default streams
    hipStreamCreate(&stream);

    hipGraph_t graph;
    hipGraphExec_t instance;
    hipGraphCreate(&graph, 0);
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    addValue<<<blocks, 1, 0, stream>>>(d_out, d_in, n, blockChunk);

    assignClearValue<<<blocks, 1, 0, stream>>>(d_out, d_in, n, blockChunk);

    hipStreamEndCapture(stream, &graph);

    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    for (size_t iter = 0; iter < k; iter++)
    {
        error = hipGraphLaunch(instance, stream);
        if (error != hipSuccess)
        {
            fprintf(stderr, "Graph launch iteration %ld failed: %s\n", iter, hipGetErrorString(error));
            printf("Error: %d\n", error);
            return;
        }
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipStreamSynchronize(stream);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    hipStreamDestroy(stream);
    hipGraphExecDestroy(instance);
    hipGraphDestroy(graph);

    // Copy the output back to the host
    error = hipMemcpyAsync(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // output is in d_in
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFreeAsync(d_in, 0);
    hipFreeAsync(d_out, 0);
}

void isingCudaGenStreams(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMallocAsync((void **)&d_in, n2 * sizeof(uint8_t), 0); // not async as we also need to copy the input
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMallocAsync((void **)&d_out, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the input from CPU to the device
    error = hipMemcpyAsync(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk;
    if (blocks > n2)
    {
        std::cout << "No need for that many blocks. Using " << n2 << " blocks" << std::endl;
        blocks = n2;
        blockChunk = 1;
    }
    else
    {
        blockChunk = n2 / blocks;                         // number of elements each block will process
        blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                          // processed per block will be as expected
    }

    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks. Using " << MAX_BLOCKS << " blocks" << std::endl;
        blocks = MAX_BLOCKS;
        blockChunk = (uint32_t)ceil((double)n2 / blocks);
    }

    funcP upFunc, downFunc, leftFunc, rightFunc, centerFunc;

    error = hipMemcpyFromSymbolAsync(&upFunc, HIP_SYMBOL(upP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of upFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbolAsync(&downFunc, HIP_SYMBOL(downP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of downFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbolAsync(&leftFunc, HIP_SYMBOL(leftP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of leftFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbolAsync(&rightFunc, HIP_SYMBOL(rightP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of rightFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbolAsync(&centerFunc, HIP_SYMBOL(centerP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of centerFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);

    hipStream_t up, down, left, right, center;
    hipStreamCreate(&up);
    hipStreamCreate(&down);
    hipStreamCreate(&left);
    hipStreamCreate(&right);
    hipStreamCreate(&center);

    for (size_t iter = 0; iter < k; iter++)
    {
        // Launch the kernel
        addValueStreams<<<blocks, 1, 0, up>>>(d_out, d_in, n, blockChunk, upFunc);
        addValueStreams<<<blocks, 1, 0, down>>>(d_out, d_in, n, blockChunk, downFunc);
        addValueStreams<<<blocks, 1, 0, left>>>(d_out, d_in, n, blockChunk, leftFunc);
        addValueStreams<<<blocks, 1, 0, right>>>(d_out, d_in, n, blockChunk, rightFunc);
        addValueStreams<<<blocks, 1, 0, center>>>(d_out, d_in, n, blockChunk, centerFunc);
        error = hipGetLastError(); // Since no error was returned from all the previous cuda calls,
                                    // the last error must be from the kernel launches

        if (error != hipSuccess)
        {
            fprintf(stderr, "Add Kernel launch failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
        }

        assignClearValueStreams<<<blocks, 1>>>(d_out, d_in, n, blockChunk); // no sync needed because default stream is synchronous to the others
        error = hipGetLastError();                                         // Since no error was returned from all the previous cuda calls,
                                                                            // the last error must be from the kernel launches
        if (error != hipSuccess)
        {
            fprintf(stderr, "Assign Kernel launch failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
        }
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipStreamSynchronize(0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    hipStreamDestroy(up);
    hipStreamDestroy(down);
    hipStreamDestroy(left);
    hipStreamDestroy(right);
    hipStreamDestroy(center);

    // Copy the output back to the host
    error = hipMemcpyAsync(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // output is in d_in
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFreeAsync(d_in, 0);
    hipFreeAsync(d_out, 0);
}

void isingCudaGenGraphStreams(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMalloc((void **)&d_in, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMalloc((void **)&d_out, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the input from CPU to the device
    error = hipMemcpy(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk;
    if (blocks > n2)
    {
        std::cout << "No need for that many blocks. Using " << n2 << " blocks" << std::endl;
        blocks = n2;
        blockChunk = 1;
    }
    else
    {
        blockChunk = n2 / blocks;                         // number of elements each block will process
        blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                          // processed per block will be as expected
    }

    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks. Using " << MAX_BLOCKS << " blocks" << std::endl;
        blocks = MAX_BLOCKS;
        blockChunk = (uint32_t)ceil((double)n2 / blocks);
    }

    funcP upFunc, downFunc, leftFunc, rightFunc, centerFunc;

    error = hipMemcpyFromSymbol(&upFunc, HIP_SYMBOL(upP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of upFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbol(&downFunc, HIP_SYMBOL(downP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of downFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbol(&leftFunc, HIP_SYMBOL(leftP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of leftFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbol(&rightFunc, HIP_SYMBOL(rightP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of rightFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMemcpyFromSymbol(&centerFunc, HIP_SYMBOL(centerP), sizeof(funcP));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of centerFunc failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);

    // Create the graph
    hipGraph_t graph;
    hipGraphExec_t instance;
    hipGraphNode_t addValueNodes[5], assignClearValueNode;
    hipKernelNodeParams kernelNodeParams[5] = {0};
    hipKernelNodeParams assignClearValueNodeParams = {0};

    // Create an empty graph
    error = hipGraphCreate(&graph, 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Graph creation failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Parameters for the addValue kernels
    void *addValueArgs[5] = {(void *)&d_out, (void *)&d_in, (void *)&n, (void *)&blockChunk, (void *)&upFunc}; // Repeat for other functions

    // Create nodes for the addValue kernel launches
    for (int i = 0; i < 5; ++i)
    {
        kernelNodeParams[i].func = (void *)addValueStreams; // Assuming addValue is a global function
        kernelNodeParams[i].gridDim = blocks;
        kernelNodeParams[i].blockDim = 1;
        kernelNodeParams[i].sharedMemBytes = 0;
        kernelNodeParams[i].extra = NULL;

        switch (i)
        {
        case 0:
            addValueArgs[4] = (void *)&upFunc;
            break;
        case 1:
            addValueArgs[4] = (void *)&downFunc;
            break;
        case 2:
            addValueArgs[4] = (void *)&leftFunc;
            break;
        case 3:
            addValueArgs[4] = (void *)&rightFunc;
            break;
        case 4:
            addValueArgs[4] = (void *)&centerFunc;
            break;
        }
        kernelNodeParams[i].kernelParams = addValueArgs;

        error = hipGraphAddKernelNode(&addValueNodes[i], graph, NULL, 0, &kernelNodeParams[i]);
        if (error != hipSuccess)
        {
            fprintf(stderr, "addValue kernel %d node addition failed: %s\n", i, hipGetErrorString(error));
            printf("Error: %d\n", error);
            return;
        }
    }

    // Parameters for the assignClearValue kernel
    void *assignClearValueArgs[4] = {(void *)&d_out, (void *)&d_in, (void *)&n, (void *)&blockChunk};

    assignClearValueNodeParams.func = (void *)assignClearValueStreams;
    assignClearValueNodeParams.gridDim = blocks;
    assignClearValueNodeParams.blockDim = 1;
    assignClearValueNodeParams.sharedMemBytes = 0;
    assignClearValueNodeParams.kernelParams = assignClearValueArgs;
    assignClearValueNodeParams.extra = NULL;

    // Create node for the assignClearValue kernel launch
    error = hipGraphAddKernelNode(&assignClearValueNode, graph, NULL, 0, &assignClearValueNodeParams);
    if (error != hipSuccess)
    {
        fprintf(stderr, "assign kernel node addition failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Set dependencies (assuming assignClearValue depends on all addValue kernels)
    for (int i = 0; i < 5; ++i)
    {
        error = hipGraphAddDependencies(graph, &addValueNodes[i], &assignClearValueNode, 1);
        if (error != hipSuccess)
        {
            fprintf(stderr, "add depedency failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
            return;
        }
    }

    // Instantiate and launch the graph
    error = hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Graph instantiation failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    for (size_t iter = 0; iter < k; iter++)
    {

        error = hipGraphLaunch(instance, 0); // Assuming stream 0 for simplicity
        if (error != hipSuccess)
        {
            fprintf(stderr, "Graph launch iteration %ld failed: %s\n", iter, hipGetErrorString(error));
            printf("Error: %d\n", error);
            return;
        }
    }

    // Synchronize
    error = hipStreamSynchronize(0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Clean up
    hipGraphExecDestroy(instance);
    hipGraphDestroy(graph);

    // Copy the output back to the host
    error = hipMemcpy(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // output is in d_in
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFree(d_in);
    hipFree(d_out);
}