#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaThreads.cuh"

__global__ void isingModel(uint8_t *out, uint8_t *in, const size_t n, const uint32_t k, const uint32_t blockChunk, uint32_t *blockCounter)
{
    size_t n2 = n * n;
    size_t threadChunk = blockChunk / blockDim.x;
    size_t start = blockIdx.x * blockChunk + threadIdx.x * threadChunk;
    size_t end = threadIdx.x == blockDim.x - 1 ? start + threadChunk + (blockChunk - blockDim.x * threadChunk) : start + threadChunk;
    if (end > n2)
        end = n2;

    // printf("gridDim.x: %d, blockIdx.x: %d, threadChunk: %d, start: %ld, end: %ld\n", gridDim.x, blockIdx.x, threadChunk, start, end);
    printf("threadIdx.x: %d, gridDim.x: %d, start: %ld, end: %ld\n", threadIdx.x, gridDim.x, start, end);

    for (size_t iter = 0; iter < k; iter++)
    {
        for (size_t i = start; i < end; i++)
        {
            uint8_t sum = in[i] + in[(i + n) % n2] + in[(i - n + n2) % n2] + in[(i + 1) % n] + in[(i - 1 + n) % n]; // operate in shared, sync tricky
            out[i] = sum > 2;                                                                                       // if majority is true (sum in [3,5]), out is true
        }

        if (threadIdx.x == 0)
        {
            atomicAdd(blockCounter, 1);

            while (*blockCounter < gridDim.x && *blockCounter != 0) // if blockCounter is 0, then all blocks have finished and one has initialized the counter to 0
                __threadfence_block();                              // Ensure I have the latest value of blockCounter

            *blockCounter = 0;
        }
        __syncthreads();
        memcpy(&in[start], &out[start], sizeof(uint8_t) * (end - start));
    }
}

void isingCuda(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const size_t n, const uint32_t k, uint32_t blocks, uint32_t threads)
{
    // check if in vector has the right dimensions
    if (in.size() != n * n)
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n * n);

    // Allocate memory on the device
    uint8_t *d_in, *d_out;
    hipMalloc((void **)&d_in, n * n * sizeof(uint8_t));
    hipMalloc((void **)&d_out, n * n * sizeof(uint8_t));

    uint32_t *blockCounter;
    hipMalloc((void **)&blockCounter, sizeof(uint32_t));
    hipMemset(&blockCounter, 0, sizeof(uint32_t));

    // Copy the input to the device
    hipMemcpy(d_in, in.data(), n * n * sizeof(uint8_t), hipMemcpyHostToDevice);

    size_t n2 = n * n;
    uint32_t blockChunk = n2 / blocks;
    blocks = blocks * blockChunk == n2 ? blocks : blocks++;

    if (threads > MAX_THREADS_PER_BLOCK)
    {
        std::cout << "Error: too many threads per block. Using 1024 threads per block" << std::endl;
        threads = MAX_THREADS_PER_BLOCK;
    }

    // Launch the kernel
    isingModel<<<blocks, threads>>>(d_out, d_in, n, k, blockChunk, blockCounter);
    hipDeviceSynchronize();

    // Copy the output back to the host
    hipMemcpy(out.data(), d_out, n * n * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(blockCounter);
}