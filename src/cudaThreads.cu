#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaThreads.cuh"

__global__ void isingModel(uint8_t *out, uint8_t *in, const size_t n, const uint32_t k, const uint32_t blockChunk, uint32_t *blockCounter)
{
    size_t n2 = n * n;
    size_t threadChunk = blockChunk / blockDim.x;
    size_t start = blockIdx.x * blockChunk + threadIdx.x * threadChunk;
    size_t end = threadIdx.x == blockDim.x - 1 ? start + threadChunk + (blockChunk - blockDim.x * threadChunk) : start + threadChunk;
    if (end > n2)
        end = n2;

    __shared__ uint8_t s[MAX_SHARED_PER_BLOCK];
    uint8_t *s_in = s;
    uint8_t *s_out = &s[blockChunk];
    memcpy(&s_in[start], &in[start], sizeof(uint8_t) * (end - start));

    // printf("gridDim.x: %d, blockIdx.x: %d, threadChunk: %d, start: %ld, end: %ld\n", gridDim.x, blockIdx.x, threadChunk, start, end);
    // printf("threadIdx.x: %d, gridDim.x: %d, start: %ld, end: %ld\n", threadIdx.x, gridDim.x, start, end);

    for (size_t iter = 0; iter < k; iter++)
    {
        __syncthreads();
        for (size_t i = start; i < end; i++)
        {
            uint8_t sum = s_in[i] + s_in[(i + n) % n2] + s_in[(i - n + n2) % n2] + s_in[(i + 1) % n] + s_in[(i - 1 + n) % n]; // operate in shared, sync tricky
            s_out[i] = sum > 2;                                                                                               // if majority is true (sum in [3,5]), out is true
        }

        if (threadIdx.x == 0)
        {
            atomicAdd(blockCounter, 1);

            while (*blockCounter < gridDim.x && *blockCounter != 0) // if blockCounter is 0, then all blocks have finished and one has initialized the counter to 0
                __threadfence_block();                              // Ensure I have the latest value of blockCounter

            *blockCounter = 0;
        }
        __syncthreads();
        memcpy(&s_in[start], &s_out[start], sizeof(uint8_t) * (end - start));
    }

    memcpy(&out[start], &s_out[start], sizeof(uint8_t) * (end - start));
}

void isingCuda(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const size_t n, const uint32_t k, uint32_t blocks, uint32_t threads)
{
    // check if in vector has the right dimensions
    if (in.size() != n * n)
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n * n);

    // Allocate memory on the device
    uint8_t *d_in, *d_out;
    hipError_t error = hipMalloc((void **)&d_in, n * n * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        // Additional error handling if needed
    }
    error = hipMalloc((void **)&d_out, n * n * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        // Additional error handling if needed
    }

    uint32_t *blockCounter;
    error = hipMalloc((void **)&blockCounter, sizeof(uint32_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        // Additional error handling if needed
    }
    error = hipMemset(blockCounter, 0, sizeof(uint32_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memset of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        // Additional error handling if needed
    }

    // Copy the input to the device
    error = hipMemcpy(d_in, in.data(), n * n * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        // Additional error handling if needed
    }

    size_t n2 = n * n;
    uint32_t blockChunk = n2 / blocks;
    blocks = blocks * blockChunk == n2 ? blocks : blocks++;

    if (threads > MAX_THREADS_PER_BLOCK)
    {
        std::cout << "Error: too many threads per block. Using 1024 threads per block" << std::endl;
        threads = MAX_THREADS_PER_BLOCK;
    }

    // Launch the kernel
    isingModel<<<blocks, threads>>>(d_out, d_in, n, k, blockChunk, blockCounter);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        printf("Error: %d\n", cudaStatus);
        // Additional error handling if needed
    }
    hipDeviceSynchronize();

    // Copy the output back to the host
    hipMemcpy(out.data(), d_out, n * n * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(blockCounter);
}