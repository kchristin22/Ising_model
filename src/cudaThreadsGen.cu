#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_cooperative_groups.h>
#include "cudaThreads.cuh"

__global__ void addValue(uint8_t *out, const uint8_t *in, const size_t n)
{
    size_t n2 = n * n;
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n2)
        return;

    size_t up = (index - n + n2) % n2;
    size_t down = (index + n) % n2;
    size_t row = index / n;
    size_t left = row * n + (index - 1 + n) % n;
    size_t right = row * n + (index + 1) % n;
    out[index] = (in[index] + in[up] + in[down] + in[left] + in[right]) > 2;
}

__global__ void assignClearValue(uint8_t *out, uint8_t *in, const size_t n)
{
    size_t n2 = n * n;
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n2)
        return;

    in[index] = out[index]; // swap the pointers and assign the final value of this iteration
}

void isingCudaGen(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMallocAsync((void **)&d_in, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMallocAsync((void **)&d_out, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the input from CPU to the device
    error = hipMemcpyAsync(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Calculate the number of blocks and threads needed to assign a single element to each thread
    uint32_t blocks = (uint32_t)ceil((double)n2 / MAX_THREADS_PER_BLOCK);
    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks needed for this input array" << std::endl;
        return;
    }
    uint32_t threads = (uint32_t)ceil((double)n2 / blocks); // distribute the elements evenly among the blocks

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);

    for (size_t iter = 0; iter < k; iter++)
    {
        // Launch the kernel
        addValue<<<blocks, threads>>>(d_out, d_in, n);
        error = hipGetLastError(); // Since no error was returned from all the previous cuda calls,
                                    // the last error must be from the kernel launches

        if (error != hipSuccess)
        {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
        }

        // Swap the pointers to prepare for the next iteration
        uint8_t *temp = d_in;
        d_in = d_out;
        d_out = temp;
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipStreamSynchronize(0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the output back to the host
    error = hipMemcpyAsync(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // output is in d_in
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFreeAsync(d_in, 0);
    hipFreeAsync(d_out, 0);
}

void isingCudaGenGraph(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMallocAsync((void **)&d_in, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMallocAsync((void **)&d_out, n2 * sizeof(uint8_t), 0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the input from CPU to the device
    error = hipMemcpyAsync(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Calculate the number of blocks and threads needed to assign a single element to each thread
    uint32_t blocks = (uint32_t)ceil((double)n2 / MAX_THREADS_PER_BLOCK);
    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks needed for this input array" << std::endl;
        return;
    }
    uint32_t threads = (uint32_t)ceil((double)n2 / blocks); // distribute the elements evenly among the blocks

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);

    hipStream_t stream; // stream capture is only supported on non-default streams
    hipStreamCreate(&stream);

    hipGraph_t graph;
    hipGraphExec_t instance;
    hipGraphCreate(&graph, 0);
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    for (size_t iter = 0; iter < k; iter++)
    {
        addValue<<<blocks, threads, 0, stream>>>(d_out, d_in, n);

        // use the kernel equivalent of pointer swapping to take advantage of graph instantiation
        assignClearValue<<<blocks, threads, 0, stream>>>(d_out, d_in, n);
    }

    hipStreamEndCapture(stream, &graph);

    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    error = hipGraphLaunch(instance, stream);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Graph launch failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipStreamSynchronize(stream);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    hipStreamDestroy(stream);
    hipGraphExecDestroy(instance);
    hipGraphDestroy(graph);

    // Copy the output back to the host
    error = hipMemcpyAsync(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // output is in d_in
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFreeAsync(d_in, 0);
    hipFreeAsync(d_out, 0);
}