#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaThreadsShared.cuh"

__global__ void isingModelGen(uint8_t *out, uint8_t *in, const size_t n, const uint32_t blockChunk, uint32_t *blockCounter)
{
    size_t n2 = n * n;
    size_t threadChunk = blockChunk / blockDim.x; // not ceil to ensure that the total number of elements processed per block is not greater than blockChunk
    size_t blockChunkStart = blockIdx.x * blockChunk;
    size_t start = blockChunkStart + threadIdx.x * threadChunk;
    size_t end = threadIdx.x == blockDim.x - 1 ? start + threadChunk + (blockChunk - blockDim.x * threadChunk) : start + threadChunk; // last thread of the block processes the remaining elements
    if (end > n2)
        end = n2;

    __shared__ uint8_t s[MAX_SHARED_PER_BLOCK]; // max shared memory macro is defined in bytes which is the size of each element
    uint8_t *s_in = s;
    uint8_t *s_out = &s[blockChunk]; // s_in has blockChunk elements

    // printf("blockChunk: %d, start: %ld, end: %ld\n", blockChunk, start, end);
    // printf("blockChunkStart: %ld\n", blockChunkStart);

    if (blockChunkStart == 0)
        blockChunkStart = end; // needed for the indexing of the shared memory
    // printf("start: %ld, blockChunkStart: %ld, start mod blockChunkStart: %ld\n", start, blockChunkStart, start % blockChunkStart);

    memcpy(&s_in[start % blockChunkStart], &in[start], sizeof(uint8_t) * (end - start)); // needed for the inter-block communication

    __syncthreads(); // other threads of the block wait for thread 0

    for (size_t i = start; i < end; i++)
    {
        // printf("up: %d, down: %d, left: %d, right: %d\n", (i - n + n2) % n2, (i + n) % n2, (i / n * n) + (i - 1 + n) % n2, (i / n * n) + (i + 1) % n2);
        size_t up = (i - n + n2) % n2;
        size_t in_up = up >= start && up < end ? s_in[up % blockChunkStart] : in[up];
        size_t down = (i + n) % n2;
        size_t in_down = down >= start && down < end ? s_in[down % blockChunkStart] : in[down];
        size_t row = i / n;
        size_t left = row * n + (i - 1 + n) % n;
        size_t in_left = left >= start && left < end ? s_in[left % blockChunkStart] : in[left];
        size_t right = row * n + (i + 1) % n;
        size_t in_right = right >= start && right < end ? s_in[right % blockChunkStart] : in[right];
        uint8_t sum = s_in[i % blockChunkStart] + in_up + in_down + in_left + in_right;
        s_out[i % blockChunkStart] = sum > 2; // assign the majority
    }

    memcpy(&out[start], &s_out[start % blockChunkStart], sizeof(uint8_t) * (end - start)); // copy the result to the global memory (the last swap is not needed)
}

void isingCudaGen(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks, uint32_t threads)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMalloc((void **)&d_in, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMalloc((void **)&d_out, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t *blockCounter; // used to sync the blocks
    error = hipMalloc((void **)&blockCounter, sizeof(uint32_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }
    error = hipMemset(blockCounter, 0, sizeof(uint32_t)); // initialize block counter to 0
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memset of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }

    // Copy the input from CPU to the device
    error = hipMemcpy(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk = n2 / blocks;                // number of elements each block will process
    blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                      // processed per block will be as expected

    if (blockChunk > MAX_SHARED_PER_BLOCK)
    {
        std::cout << "Error: too many elements per block. Use more blocks." << std::endl;
        return;
    }

    if (threads > MAX_THREADS_PER_BLOCK)
    {
        std::cout << "Error: too many threads per block. Using " << MAX_THREADS_PER_BLOCK << " threads per block" << std::endl;
        threads = MAX_THREADS_PER_BLOCK;
    }

    for (size_t iter = 0; iter < k; iter++)
    {
        // Launch the kernel
        isingModelGen<<<blocks, threads>>>(d_out, d_in, (size_t)sqrt(n2), blockChunk, blockCounter);
        error = hipGetLastError(); // Since no error was returned from all the previous cuda calls,
                                    // the last error must be from the kernel launch
        if (error != hipSuccess)
        {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
        }

        // Wait for the kernel to finish to avoid exiting the program prematurely
        error = hipDeviceSynchronize();
        if (error != hipSuccess)
        {
            fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
            printf("Error: %d\n", error);
            return;
        }

        // swap the pointers
        uint8_t *temp = d_in;
        d_in = d_out;
        d_out = temp;

        // printf("d_out: %p, d_in: %p\n", d_out, d_in);
    }

    // Copy the output back to the host
    error = hipMemcpy(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost); // the last swap is not needed, so d_in is the final result
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    // Free the memory on the device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(blockCounter);
}