#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "cudaBlocks.cuh"

__global__ void isingModelBlocks(uint8_t *out, uint8_t *in, const size_t n, const uint32_t k, const uint32_t blockChunk, uint32_t *blockCounter, bool *allBlocksFinished)
{
    size_t n2 = n * n;
    size_t start = blockIdx.x * blockChunk;
    size_t end = start + blockChunk;
    if (end > n2)
        end = n2;

    cooperative_groups::grid_group g = cooperative_groups::this_grid(); // used for the inter-block communication

    for (size_t iter = 0; iter < k; iter++)
    {
        for (size_t i = start; i < end; i++)
        {
            size_t up = (i - n + n2) % n2;
            size_t down = (i + n) % n2;
            size_t row = i / n;
            size_t left = row * n + (i - 1 + n) % n;
            size_t right = row * n + (i + 1) % n;
            uint8_t sum = in[i] + in[up] + in[down] + in[left] + in[right];
            out[i] = sum > 2; // assign the majority
        }

        // sync the running blocks before swapping the pointers
        g.sync();

        /* Without cooperative groups version

        atomicAdd(blockCounter, 1); // this block has finished
        __threadfence();            // ensure that threads reading the value of blockCounter from now on cannot see the previous value

        *allBlocksFinished = false;
        __threadfence();
        while (!(*allBlocksFinished))
        {
            __threadfence(); // rest of the blocks load the new value of allBlocksFinished

            if (*blockCounter == gridDim.x)
            {
                *allBlocksFinished = true;
                __threadfence(); // update the value of allBlocksFinished
            }
        }
        *blockCounter = 0; // re-set this block's value to 0
        __threadfence();

        */

        // swap the pointers
        uint8_t *temp = in;
        in = out;
        out = temp;
    }
}

void isingCuda(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMalloc((void **)&d_in, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMalloc((void **)&d_out, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t *blockCounter; // used to sync the blocks
    error = hipMalloc((void **)&blockCounter, sizeof(uint32_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }
    error = hipMemset(blockCounter, 0, sizeof(uint32_t)); // initialize block counter to 0
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memset of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }

    // Copy the input from CPU to the device
    error = hipMemcpy(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk;
    if (blocks > n2)
    {
        std::cout << "No need for that many blocks. Using " << n2 << " blocks" << std::endl;
        blocks = n2;
        blockChunk = 1;
    }
    else
    {
        blockChunk = n2 / blocks;                         // number of elements each block will process
        blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                          // processed per block will be as expected
    }

    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks. Using " << MAX_BLOCKS << " blocks" << std::endl;
        blocks = MAX_BLOCKS;
        blockChunk = (uint32_t)ceil((double)n2 / blocks);
    }

    // Allocate memory for the flag that indicates if all blocks have finished
    bool *allBlocksFinished;
    error = hipMalloc((void **)&allBlocksFinished, sizeof(bool));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of allBlocksFinished failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }
    error = hipMemset(allBlocksFinished, false, sizeof(bool));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memset of allBlocksFinished failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }

    // Set arguments for the kernel
    size_t n = (size_t)sqrt(n2);
    void *kernelArgs[] = {&d_out, &d_in, &n, (void *)&k, &blockChunk, &blockCounter, &allBlocksFinished};

    // Launch the kernel
    error = hipLaunchCooperativeKernel((void *)isingModelBlocks, blocks, 1, (void **)kernelArgs);

    /* Or if your device doesn't support cooperative groups

    isingModelBlocks<<<blocks, 1>>>(d_out, d_in, (size_t)sqrt(n2), k, blockChunk, blockCounter, allBlocksFinished);
    error = hipGetLastError(); // Since no error was returned from all the previous cuda calls,
                                // the last error must be from the kernel launch

    */

    if (error != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the output back to the host
    if (k % 2 == 0)
        error = hipMemcpy(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost);
    else
        error = hipMemcpy(out.data(), d_out, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(blockCounter);
    hipFree(allBlocksFinished);
}