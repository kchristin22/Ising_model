#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaBlocks.cuh"

__global__ void isingModelBlocks(uint8_t *out, uint8_t *in, const size_t n, const uint32_t k, const uint32_t blockChunk, uint32_t *blockCounter)
{
    size_t n2 = n * n;
    size_t start = blockIdx.x * blockChunk;
    size_t end = start + blockChunk;
    if (end > n2)
        end = n2;

    for (size_t iter = 0; iter < k; iter++)
    {
        for (size_t i = start; i < end; i++)
        {
            size_t up = (i - n + n2) % n2;
            size_t down = (i + n) % n2;
            size_t row = i / n;
            size_t left = row * n + (i - 1 + n) % n;
            size_t right = row * n + (i + 1) % n;
            uint8_t sum = in[i] + in[up] + in[down] + in[left] + in[right];
            out[i] = sum > 2; // assign the majority
        }

        // sync the running blocks before swapping the pointers
        atomicAdd(blockCounter, 1); // this block has finished
        __threadfence();            // ensure that threads reading the value of blockCounter from now on cannot see the previous value

        while (*blockCounter < gridDim.x && *blockCounter != 0)
            ; // if blockCounter is 0, then all blocks have finished and one has initialized the counter to 0

        *blockCounter = 0;

        // swap the pointers
        uint8_t *temp = in;
        in = out;
        out = temp;
    }
}

void isingCuda(std::vector<uint8_t> &out, std::vector<uint8_t> &in, const uint32_t k, uint32_t blocks)
{
    size_t n2 = in.size();
    // check if `in` vector has a perfect square size
    if (ceil(sqrt(n2)) != floor(sqrt(n2)))
    {
        std::cout << "Error: input vector has wrong dimensions" << std::endl;
        return;
    }
    out.resize(n2);

    // Allocate memory on the device (GPU)
    uint8_t *d_in, *d_out;
    hipError_t error = hipMalloc((void **)&d_in, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }
    error = hipMalloc((void **)&d_out, n2 * sizeof(uint8_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of d_out failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t *blockCounter; // used to sync the blocks
    error = hipMalloc((void **)&blockCounter, sizeof(uint32_t));
    if (error != hipSuccess)
    {
        fprintf(stderr, "Malloc of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }
    error = hipMemset(blockCounter, 0, sizeof(uint32_t)); // initialize block counter to 0
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memset of blockCounter failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }

    // Copy the input from CPU to the device
    error = hipMemcpy(d_in, in.data(), n2 * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of d_in failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    uint32_t blockChunk = n2 / blocks;                // number of elements each block will process
    blocks = (uint32_t)ceil((double)n2 / blockChunk); // the actual number of blocks may change but the total number of elements
                                                      // processed per block will be as expected

    if (blocks > MAX_BLOCKS)
    {
        std::cout << "Error: too many blocks. Using " << MAX_BLOCKS << " blocks" << std::endl;
        blocks = MAX_BLOCKS;
        blockChunk = (uint32_t)ceil((double)n2 / blocks);
    }

    // Launch the kernel
    isingModelBlocks<<<blocks, 1>>>(d_out, d_in, (size_t)sqrt(n2), k, blockChunk, blockCounter);
    error = hipGetLastError(); // Since no error was returned from all the previous cuda calls,
                                // the last error must be from the kernel launch
    if (error != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
    }

    // Wait for the kernel to finish to avoid exiting the program prematurely
    error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Copy the output back to the host
    if (k % 2 == 0)
        error = hipMemcpy(out.data(), d_in, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost);
    else
        error = hipMemcpy(out.data(), d_out, n2 * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Memcpy of device's output to host failed: %s\n", hipGetErrorString(error));
        printf("Error: %d\n", error);
        return;
    }

    // Free the memory on the device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(blockCounter);
}